#include "hip/hip_runtime.h"


#include <stdio.h>
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"

#define N 100 /* runners*/
#define MAX 6 /* runners max speed 5 */

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

    /* we have to initialize the state */
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* speeds) {


    speeds[blockIdx.x] = hiprand(&states[blockIdx.x]) % 6;
    if (speeds[blockIdx.x] == 0)
        speeds[blockIdx.x] += 1;

}
void Delay(unsigned int time) {
    for(int j=0; j<time;j++){}
}



int main() {

   
  //vaariables
    hiprandState_t* states;
    unsigned int cpu_nums[N];
    unsigned int* gpu_nums;
    unsigned int location[N];
    unsigned int i;
    
    while (1) {
        unsigned int wait = 100000;

        /* allocate space on the GPU for the random states */
        hipMalloc((void**)&states, N * sizeof(hiprandState_t));

        /* invoke the GPU to initialize all of the random states */
        init << < N, 1 >> > (time(0), states);
        hipDeviceSynchronize();
        /* allocate an array of unsigned ints on the CPU and GPU */

        hipMalloc((void**)&gpu_nums, N * sizeof(unsigned int));

        /*  kernel to get some random numbers */
        randoms << < N, 1 >> > (states, gpu_nums);
        hipDeviceSynchronize();

        /* copy the random numbers back */
        hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);


        /* print them out */
        for (i = 0; i < N; i++) {
            location[i] = cpu_nums[i] + location[i];
            printf("%d  nolu yarismaci \t %u anlik hizi \t %u yarismaci konumu \n", i, cpu_nums[i], location[i]);
        }

        Delay(wait);

        /* free the memory we allocated for the states and numbers */
        hipFree(states);
        hipFree(gpu_nums);


    }
    
    
   return 0;
}
